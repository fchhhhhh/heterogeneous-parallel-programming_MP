#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                       \
    do {                                                    \
        hipError_t err = stmt;                             \
        if (err != hipSuccess) {                           \
            wbLog(ERROR, "Failed to run stmt ", #stmt);     \
            return -1;                                      \
        }                                                   \
    } while(0)

#define Mask_width  5
#define Mask_radius Mask_width / 2
#define TILE_WIDTH  16
#define SIZE        (TILE_WIDTH + Mask_width - 1)

//@@ INSERT CODE HERE
__global__ void convolution2D (float * I, const float * __restrict__ M, float * P,
        int channels, int width, int height)
{
    __shared__ float N_ds[SIZE][SIZE];

    int bx = blockIdx.x,  by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;

    for (int k = 0; k < channels; ++k) {
        int dest  = ty * TILE_WIDTH + tx;
        int destX = dest % SIZE;
        int destY = dest / SIZE;
        int srcY  = by * TILE_WIDTH + destY - Mask_radius;
        int srcX  = bx * TILE_WIDTH + destX - Mask_radius;
        int src   = (srcY * width + srcX) * channels + k;

        if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
            N_ds[destY][destX] = I[src];
        else
            N_ds[destY][destX] = 0.0;

        dest  = ty * TILE_WIDTH + tx + TILE_WIDTH * TILE_WIDTH;
        destY = dest / SIZE;
        destX = dest % SIZE;
        srcY  = by * TILE_WIDTH + destY - Mask_radius;
        srcX  = bx * TILE_WIDTH + destX - Mask_radius;
        src   = (srcY * width + srcX) * channels + k;

        if (destY < SIZE) {
            if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
                N_ds[destY][destX] = I[src];
            else
                N_ds[destY][destX] = 0.0;
        }
        __syncthreads();

        float accum = 0;
        for (int y = 0; y < Mask_width; ++y)
            for (int x = 0; x < Mask_width; ++x)
                accum += N_ds[ty + y][tx + x] * M[y * Mask_width + x];

        int y = by * TILE_WIDTH + ty;
        int x = bx * TILE_WIDTH + tx;
        if (y < height && x < width)
            P[(y * width + x) * channels + k] = min(max(accum, 0.0), 1.0);

        __syncthreads();
    }
}



int main (int argc, char * argv[ ])
{
    wbArg_t arg;
    int maskRows;
    int maskColumns;
    int imageChannels;
    int imageWidth;
    int imageHeight;
    char * inputImageFile;
    char * inputMaskFile;
    wbImage_t inputImage;
    wbImage_t outputImage;
    float * hostInputImageData;
    float * hostOutputImageData;
    float * hostMaskData;
    float * deviceInputImageData;
    float * deviceOutputImageData;
    float * deviceMaskData;

    arg = wbArg_read(argc, argv); /* parse the input arguments */

    inputImageFile = wbArg_getInputFile(arg, 0);
    inputMaskFile = wbArg_getInputFile(arg, 1);

    inputImage = wbImport(inputImageFile);
    hostMaskData = (float *) wbImport(inputMaskFile, &maskRows, &maskColumns);

    assert(maskRows == 5); /* mask height is fixed to 5 in this mp */
    assert(maskColumns == 5); /* mask width is fixed to 5 in this mp */

    imageWidth = wbImage_getWidth(inputImage);
    imageHeight = wbImage_getHeight(inputImage);
    imageChannels = wbImage_getChannels(inputImage);

    outputImage = wbImage_new(imageWidth, imageHeight, imageChannels);

    hostInputImageData = wbImage_getData(inputImage);
    hostOutputImageData = wbImage_getData(outputImage);

    wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

    wbTime_start(GPU, "Doing GPU memory allocation");
    hipMalloc((void **) &deviceInputImageData, imageWidth * imageHeight * imageChannels * sizeof(float));
    hipMalloc((void **) &deviceOutputImageData, imageWidth * imageHeight * imageChannels * sizeof(float));
    hipMalloc((void **) &deviceMaskData, maskRows * maskColumns * sizeof(float));
    wbTime_stop(GPU, "Doing GPU memory allocation");

    wbTime_start(Copy, "Copying data to the GPU");
    hipMemcpy(deviceInputImageData, hostInputImageData,
               imageWidth * imageHeight * imageChannels * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(deviceMaskData, hostMaskData,
               maskRows * maskColumns * sizeof(float),
               hipMemcpyHostToDevice);
    wbTime_stop(Copy, "Copying data to the GPU");

    wbTime_start(Compute, "Doing the computation on the GPU");
    //@@ INSERT CODE HERE
    dim3 dimGrid(ceil((float) imageWidth / TILE_WIDTH),
                 ceil((float) imageHeight / TILE_WIDTH));
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    convolution2D<<<dimGrid, dimBlock>>>(deviceInputImageData, deviceMaskData,
            deviceOutputImageData, imageChannels, imageWidth, imageHeight);
    wbTime_stop(Compute, "Doing the computation on the GPU");

    wbTime_start(Copy, "Copying data from the GPU");
    hipMemcpy(hostOutputImageData, deviceOutputImageData,
               imageWidth * imageHeight * imageChannels * sizeof(float),
               hipMemcpyDeviceToHost);
    wbTime_stop(Copy, "Copying data from the GPU");

    wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

    wbSolution(arg, outputImage);

    hipFree(deviceInputImageData);
    hipFree(deviceOutputImageData);
    hipFree(deviceMaskData);

    free(hostMaskData);
    wbImage_delete(outputImage);
    wbImage_delete(inputImage);

    return 0;
}